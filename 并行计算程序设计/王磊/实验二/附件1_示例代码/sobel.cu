#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#include <hip/hip_runtime.h>   
typedef unsigned char uchar;

typedef struct{
    int w;
    int h;
    unsigned char * img;
} PGM_IMG;    

//check cuda error and set cuda device
void checkErr(hipError_t err,int num);     /*check cuda errors*/
void setCudaDevice(int devNum);

//define gray-scale images
PGM_IMG read_pgm(const char * path);
void write_pgm(PGM_IMG img, const char * path);
void free_pgm(PGM_IMG img);
//edge detection for gray-scale images
PGM_IMG edge_sobel(PGM_IMG img_in);
void sobel_cpu_kernel(uchar * img_in, uchar * img_out,int img_w, int img_h);
int sobel(uchar a, uchar b, uchar c, uchar d, uchar e, uchar f);
//call edge detection
void run_cpu_sobel_test(PGM_IMG img_in);

void run_gpu_sobel_test(PGM_IMG img_in);
PGM_IMG gpu_edge_sobel(PGM_IMG img_in);
__global__ void sobel_gpu_kernel(uchar * img_in, uchar * img_out,int img_w, int img_h);
__device__ int sobel_gpu(uchar a, uchar b, uchar c, uchar d, uchar e, uchar f);
int main()
{
	int devNum=0;   
    setCudaDevice(devNum);  
    PGM_IMG img_ibuf;
//	clock_t start, finish;
    hipEvent_t start1, stop1;
    hipEventCreate(&start1);
    hipEventCreate(&stop1);
    hipEventRecord(start1,0);
//    double duration; 
//	start = clock();
    printf("Running edge detection for gray-scale images.\n\n");
	printf("---------------Read PGM-----------------\n");
    img_ibuf = read_pgm("lena.pgm");
    run_cpu_sobel_test(img_ibuf);
//	run_gpu_sobel_test(img_ibuf);
    free_pgm(img_ibuf); 
//	finish = clock();
     hipEventRecord(stop1,0);
     hipEventSynchronize(stop1);
     float time1;
     hipEventElapsedTime(&time1,start1,stop1);
     
//	duration = (double)(finish - start) ; //CLOCKS_PER_SEC=1000,����
    printf( "The time of calculating is :%f\n", time1); 

    return 0;
}
/*-----------------------------------------------------------*/
/*---------------------two useful functions------------------*/
/*----------------------do not modify------------------------*/
/*-----------------------------------------------------------*/
 void checkErr(hipError_t err,int num)     /*check cuda errors*/
{
	 if( hipSuccess != err) {  
		fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",        
                __FILE__, num-1, hipGetErrorString( err) );              
	 }
}
void setCudaDevice(int devNum)
{
	hipError_t err = hipSuccess;
	printf("\nCUDA Device #%d\n", devNum);
	hipDeviceProp_t devProp;
	hipGetDeviceProperties(&devProp, devNum);
	printf("Name:                          %s\n",  devProp.name);
	printf("Total global memory:           %u\n",  devProp.totalGlobalMem);
	printf("Major revision number:         %d\n",  devProp.major);
	printf("Minor revision number:         %d\n",  devProp.minor);
	err=hipSetDevice(devNum);
	checkErr(err,__LINE__);
}
/*-----------------------------------------------------------*/
/*----------------------read and write pgm picture-----------*/
/*----------------------do not modify------------------------*/
/*-----------------------------------------------------------*/
PGM_IMG read_pgm(const char * path){
    FILE * in_file;
    char sbuf[256]; 
    PGM_IMG result;
    int v_max;//, i;
    in_file = fopen(path, "rb");
    if (in_file == NULL){
        printf("Input file not found!\n");
        exit(1);
    }
    fscanf(in_file, "%s", sbuf); /*  Skip the magic number,P2/P5   */
    fscanf(in_file, "%d",&result.w);
    fscanf(in_file, "%d",&result.h);
    fscanf(in_file, "%d\n",&v_max);  
    printf("size:%s\n",sbuf);
    printf("Image size: %d x %d\n", result.w, result.h);
	printf("v_max:%d\n",v_max);
    
    result.img = (unsigned char *)malloc(result.w * result.h * sizeof(unsigned char));
    fread(result.img, sizeof(unsigned char), result.w*result.h, in_file); //to result.img
	printf("Read the picture succeed!\n\n");
    fclose(in_file);
    
    return result;   //PGM_IMG
}

void write_pgm(PGM_IMG img, const char * path){
    FILE * out_file;
    out_file = fopen(path, "wb");
    fprintf(out_file, "P5\n");
    fprintf(out_file, "%d %d\n255\n",img.w, img.h);
    fwrite(img.img,sizeof(unsigned char), img.w*img.h, out_file);
    fclose(out_file);
	printf("write the file............\n");
}
void free_pgm(PGM_IMG img)
{
    free(img.img);
}
/*-----------------------------------------------------------*/
/*-------------CPU sobel edge detection test-----------------*/
/*-----------------------------------------------------------*/
void run_cpu_sobel_test(PGM_IMG img_in)
{
    PGM_IMG img_obuf;
    printf("Starting CPU processing...\n");
    img_obuf = edge_sobel(img_in);  //edge detection
    printf("Edge detection of the picture!\n");
    write_pgm(img_obuf, "result_PGM.pgm");
	printf("Write new file succeed!\n\n");
    free_pgm(img_obuf);
}
/*-----------------------------------------------------------*/
/*-------------CPU sobel operator implementation-------------*/
/*-----------------------------------------------------------*/
PGM_IMG edge_sobel(PGM_IMG img_in)
{
    PGM_IMG result;
    result.w = img_in.w;
    result.h = img_in.h;
    result.img = (unsigned char *)malloc(result.w * result.h * sizeof(unsigned char));
	memset(result.img,0,result.w * result.h * sizeof(unsigned char));
    sobel_cpu_kernel(img_in.img, result.img,result.w,result.h);
    return result;
}
void sobel_cpu_kernel(uchar * img_in, uchar * img_out,int img_w, int img_h){
    for(int i=1;i<img_h-1;i++)  
    {  
        for(int j=1;j<img_w-1;j++)  
        {  
            //ͨ��ָ�����ͼ����ÿһ������  
			/*---------------------------------*/
			/*----------x1----x2----x3---------*/
			/*----------x4----x5----x6---------*/
			/*----------x7----x8----x9---------*/
			/*---------------------------------*/
			int x1, x2, x3, x4, x5, x6, x7, x8,x9;
			x1=img_in[(i-1)*img_w+(j-1)];
			x2=img_in[(i-1)*img_w+j];
			x3=img_in[(i-1)*img_w+j+1];
			x4=img_in[i*img_w+j-1];
			x5=img_in[i*img_w+j]; // never use x5
			x6=img_in[i*img_w+j+1];
			x7=img_in[(i+1)*img_w+j-1];
			x8=img_in[(i+1)*img_w+j];
			x9=img_in[(i+1)*img_w+j+1];
			int dfdy= sobel(x1, x2, x3, x7, x8, x9);
            int dfdx= sobel(x1, x4, x7, x3, x6, x9);	
			int gradient= sqrtf(dfdy*dfdy+dfdx*dfdx);
			img_out[i*img_w+j] = gradient;

		}  
    }
}
int sobel(uchar a, uchar b, uchar c, uchar d, uchar e, uchar f) {
	return ((a + 2*b + c) - (d + 2*e + f));
}
/*-----------------------------------------------------------*/
/*-------------GPU sobel edge detection test-----------------*/
/*-----------------------------------------------------------*/
void run_gpu_sobel_test(PGM_IMG img_in)
{
    PGM_IMG img_obuf;
    printf("Starting GPU processing...\n");
    img_obuf = gpu_edge_sobel(img_in);  //edge detection
    printf("Edge detection of the picture!\n");
    write_pgm(img_obuf, "result_PGM.pgm");
	printf("Write new file succeed!\n\n");
    free_pgm(img_obuf);
}

/*-----------------------------------------------------------*/
/*-------------GPU sobel operator implementation-------------*/
/*-----------------------------------------------------------*/
PGM_IMG gpu_edge_sobel(PGM_IMG img_in)
{
    PGM_IMG img_out;
    img_out.w = img_in.w;
    img_out.h = img_in.h;
	size_t size = img_out.w * img_out.h * sizeof(uchar);
    img_out.img = (uchar *)malloc(size);
	memset(img_out.img,0,size);
	
    // Allocate the device memory for result.img
    uchar *img_temp=NULL, *d_img_in = NULL,*d_img_out = NULL;

	hipError_t err = hipSuccess;
    err = hipMalloc((void **)&d_img_in, size);
    checkErr(err,__LINE__);
	err = hipMemcpy(d_img_in, img_in.img, size, hipMemcpyHostToDevice);
    checkErr(err,__LINE__);

	err = hipMalloc((void **)&d_img_out, size);
    checkErr(err,__LINE__);
	err = hipMemcpy(d_img_out, img_out.img, size, hipMemcpyHostToDevice);
    checkErr(err,__LINE__);

	dim3 threadsPerBlock(16,16);
	dim3 blocksPerGrid((img_in.w+15)/16,(img_in.h+15)/16);

    sobel_gpu_kernel<<<blocksPerGrid,threadsPerBlock>>>(d_img_in, d_img_out,img_in.w,img_in.h);
	err = hipGetLastError();

	err = hipMemcpy(img_out.img, d_img_out, size, hipMemcpyDeviceToHost);
    checkErr(err,__LINE__);
	hipFree(d_img_in);
	hipFree(d_img_out);
	//free(img_temp);
    return img_out;
}
__global__ void sobel_gpu_kernel(uchar * img_in, uchar * img_out,int img_w, int img_h)
{
	int row = blockDim.y * blockIdx.y + threadIdx.y;
	int col = blockDim.x * blockIdx.x + threadIdx.x;

    if((row >= 0) && (row < img_h) && (col >= 0) && (col < img_w))
    {
			uchar x1, x2, x3, x4, x5, x6, x7, x8,x9;
			x1=img_in[(row-1)*img_w+(col-1)];
			x2=img_in[(row-1)*img_w+col];
			x3=img_in[(row-1)*img_w+col+1];
			x4=img_in[row*img_w+col-1];
			//x5=img_in[i*img_w+j]; // never use x5
			x6=img_in[row*img_w+col+1];
			x7=img_in[(row+1)*img_w+col-1];
			x8=img_in[(row+1)*img_w+col];
			x9=img_in[(row+1)*img_w+col+1];
			int dfdy= sobel_gpu(x1, x2, x3, x7, x8, x9);
            int dfdx= sobel_gpu(x1, x4, x7, x3, x6, x9);	
			int gradient= sqrtf(dfdy*dfdy+dfdx*dfdx);
			img_out[row*img_w+col] = gradient; 
    }
}
__device__ int sobel_gpu(uchar a, uchar b, uchar c, uchar d, uchar e, uchar f){
	return ((a + 2*b + c) - (d + 2*e + f));
}
